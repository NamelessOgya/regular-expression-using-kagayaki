#include "hip/hip_runtime.h"
/* =======================================================================
 *  sandbox/nfa.cu
 *  Thompson‑NFA 正規表現マッチャ (GPU 版)
 *  -------------------------------------------------
 *  ● re2post / post2nfa など CPU 側ロジックは nfa_cpu_common.h に分離
 *  ● 文字列ごとに 1‑thread で並列マッチ
 *  ● 最大状態数 : 4096、最大クエリ数 : 1024、最大文字列長 : 1024
 *  ビルド例 :  nvcc -O3 -std=c++17 -arch=sm_80 -Isandbox sandbox/nfa.cu -o nfa.out
 * =======================================================================*/
 #include <hip/hip_runtime.h>
 #include <vector>
 #include <unordered_map>
 #include <iostream>
 #include <cstring>
 #include "nfa_cpu_common.h"
 
 /* --------------------------- グローバル ------------------------------ */
 int   nstate      = 0;                       // CPU 側で状態数をインクリメント
 State matchstate  = { Match, nullptr, nullptr, 0 };
 
 /* --------------------------- GPU 用構造体 --------------------------- */
 struct GPUState {
     int c;
     int out;
     int out1;
     int lastlist;      // GPU カーネルでは未使用（0 固定）
 };
 
 /* -------------------- NFA を線形化するユーティリティ ----------------- */
 static std::vector<State*>            order;     // DFS 順
 static std::unordered_map<State*,int> idx_map;   // State* → 連番
 
 void gather_states(State* s)
 {
     if (!s || idx_map.count(s)) return;
     int id = static_cast<int>(order.size());
     idx_map[s] = id;
     order.push_back(s);
 
     if (s->out ) gather_states(s->out );
     if (s->out1) gather_states(s->out1);
 }
 inline int idx_of(State* s) { return s ? idx_map.at(s) : -1; }
 
 /* ---------------------- GPU カーネル補助関数 ------------------------- */
 __device__ __forceinline__
 void add_state(int* list, int& n, const GPUState* d_states,
                int idx, int list_id)
 {
     if (idx < 0) return;
     const GPUState& st = d_states[idx];
     if (st.lastlist == list_id) return;          // visited
     // Split は ε 遷移を辿る
     if (st.c == Split) {
         add_state(list, n, d_states, st.out , list_id);
         add_state(list, n, d_states, st.out1, list_id);
     } else {
         list[n++] = idx;
     }
 }
 
 __global__ void match_kernel(const GPUState* d_states,
                              const char* d_texts,
                              const int*  d_off,
                              const int*  d_len,
                              int n_strings,
                              int* d_res)
 {
     int tid = blockIdx.x * blockDim.x + threadIdx.x;
     if (tid >= n_strings) return;
 
     const char* str = d_texts + d_off[tid];
     int   len       = d_len[tid];
 
     /* スレッドローカルの NFA ワーク領域 */
     __shared__ int   list_id_shared;            // ブロック共有 list_id
     if (threadIdx.x == 0) list_id_shared = 1;
     __syncthreads();
 
     int list_id = list_id_shared;
     const int MAX_STATE = 4096;
     int clist[MAX_STATE], nlist[MAX_STATE];
     int n_c = 0, n_n = 0;
 
     add_state(clist, n_c, d_states, 0, list_id);   // 0: start
 
     for (int pos = 0; pos < len; ++pos) {
         ++list_id;
         char ch = str[pos];
         n_n = 0;
         for (int i = 0; i < n_c; ++i) {
             const GPUState& st = d_states[clist[i]];
             if (st.c == static_cast<unsigned char>(ch))
                 add_state(nlist, n_n, d_states, st.out, list_id);
         }
         // swap
         n_c = n_n;
         for (int i = 0; i < n_c; ++i) clist[i] = nlist[i];
     }
     // accept?
     int matched = 0;
     for (int i = 0; i < n_c && !matched; ++i)
         if (d_states[clist[i]].c == Match) matched = 1;
 
     d_res[tid] = matched;
 }
 
 /* -------------------------------------------------------------------- */
 int main(int argc, char** argv)
 {
     if (argc < 3) {
         std::cerr << "usage: nfa regexp string...\n";
         return 1;
     }
     /* ---------- 1. CPU で NFA 構築 ---------- */
     char* post = re2post(argv[1]);
     if (!post) { std::cerr << "bad regexp\n"; return 1; }
 
     State* start = post2nfa(post);
     if (!start) { std::cerr << "post2nfa failed\n"; return 1; }
 
     gather_states(start);                        // order[] に DFS
 
     /* ---------- 2. GPUState 配列作成 ---------- */
     std::vector<GPUState> h_states(order.size());
     for (size_t i = 0; i < order.size(); ++i) {
         State* s = order[i];
         h_states[i] = { s->c,
                         idx_of(s->out),
                         idx_of(s->out1),
                         0 };
     }
 
     /* ---------- 3. 文字列バッチ準備 ---------- */
     const int n_inputs = argc - 2;
     std::vector<int>  h_off(n_inputs), h_len(n_inputs);
     size_t total_bytes = 0;
     for (int i = 0; i < n_inputs; ++i) {
         h_off[i] = total_bytes;
         h_len[i] = static_cast<int>(strlen(argv[i+2]));
         total_bytes += h_len[i];
     }
     std::vector<char> h_texts(total_bytes);
     for (int i = 0; i < n_inputs; ++i)
         memcpy(h_texts.data() + h_off[i], argv[i+2], h_len[i]);
 
     /* ---------- 4. デバイスメモリ確保 ---------- */
     GPUState* d_states;  hipMalloc(&d_states, h_states.size()*sizeof(GPUState));
     char*     d_texts;   hipMalloc(&d_texts,  total_bytes);
     int*      d_off;     hipMalloc(&d_off,    n_inputs*sizeof(int));
     int*      d_len;     hipMalloc(&d_len,    n_inputs*sizeof(int));
     int*      d_res;     hipMalloc(&d_res,    n_inputs*sizeof(int));
 
     hipMemcpy(d_states, h_states.data(), h_states.size()*sizeof(GPUState),
                hipMemcpyHostToDevice);
     hipMemcpy(d_texts , h_texts.data() , total_bytes, hipMemcpyHostToDevice);
     hipMemcpy(d_off   , h_off.data()   , n_inputs*sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(d_len   , h_len.data()   , n_inputs*sizeof(int), hipMemcpyHostToDevice);
 
     /* ---------- 5. カーネル起動 ---------- */
     int threads = 256;
     int blocks  = (n_inputs + threads - 1) / threads;
     match_kernel<<<blocks, threads>>>(d_states,
                                       d_texts, d_off, d_len,
                                       n_inputs, d_res);
     hipDeviceSynchronize();
 
     /* ---------- 6. 結果取得・出力 ---------- */
     std::vector<int> h_res(n_inputs);
     hipMemcpy(h_res.data(), d_res, n_inputs*sizeof(int), hipMemcpyDeviceToHost);
 
     for (int i = 0; i < n_inputs; ++i)
        //  if (h_res[i]) std::cout << argv[i+2] << '\n';
        if (h_res[i]) printf("%s\n", argv[i+2]);
 
     /* ---------- 7. 後始末 ---------- */
     hipFree(d_states); hipFree(d_texts);
     hipFree(d_off);    hipFree(d_len); hipFree(d_res);
     
    
     return 0;
 }
 